#include "hip/hip_runtime.h"
/*
SCM boundary searching, SCM is performed
by sweeping the whole domain without
subdivision.
By: Free Xiong; 2015-03-05
*/
#include "hip/hip_runtime.h"
#include <cstdlib>
#include "cell.cuh"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <stdio.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

using namespace std;

#define NumberOfBlock 256
#define ThreadPerBlock 512

//sop based scm for stability boundary finding
__global__ void scm_sop(long tot_cells, int *N, float *lb, float *ub, unsigned long long *img){
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	long i,j;
	float h[D],xcs[D],xcn[D],ximg[D],gradf[D],norm;
	float fcs,fcn,df;
	int zcs[D],zcn[D],zimg[D];
	unsigned long long neighbours[pop];
	unsigned long long cs,cn;
	bool flag,boundary;
	float t,hmax,hmin;

	for(j=0;j<D;j++)
		h[j] = (ub[j]-lb[j])/(float)N[j];

	hmax = h[0];
	hmin = h[0];
	for(j=1;j<D;j++){
		if(h[j]>hmax)
			hmax = h[j];
		if(h[j]<hmin)
			hmin = h[j];
	}

	i = id;
	while(i<tot_cells){
		cs = i+1;
		celltoz(zcs,cs,N);
		ztox(xcs,zcs,h,lb);
		fcs = fp(xcs);
		flag = true;
		df = 1000.f;

		//examine neighbours, pick the steepest descent as image cell
		neighbour_finder(neighbours,N,cs);
		for(j=0;j<pop;j++){
			if(neighbours[j]==0)
				continue;
			cn = neighbours[j];
			celltoz(zcn,cn,N);
			ztox(xcn,zcn,h,lb);
			fcn = fp(xcn);
			if(fcn<fcs && fcn-fcs<df){
				df = fcn-fcs;
				img[i] = cn;
				flag = false;
			}
		}

		//we take the boundary cell as absorbing cell
		boundary = false;
		for(j=0;j<D;j++){
			if(zcs[j]==1 || zcs[j]==N[j]){
				boundary = true;
				break;
			}
		}
		if(boundary&&flag)
			img[i] = 0; //mark as sink cell
		else if(!boundary&&flag){
		//	//use gradient descend as secondary criteria
		//	f(gradf,xcs);
		//	norm = 0.f;
		//	for(j=0;j<D;j++)
		//		norm+=gradf[j]*gradf[j];
		//	norm = sqrtf(norm);

		//	t = hmax; //step size
		//	while(true){
		//		for(j=0;j<D;j++)
		//			ximg[j] = xcs[j]-t*gradf[j]/norm;
		//		if(fp(ximg)<fcs){
		//			xtoz(zimg,ximg,h,lb);
		//			img[i] = ztocell(zimg,N);
		//			break;
		//		}
		//		else if(t<hmin/2.f){
		//			img[i] = cs; //absorbing
		//			break;
		//		}
		//		else if(fp(ximg)>=fcs && t>=hmin/2.f)
		//			t/=2.f;
		//	}

			//use fminsearch as secondary criteria
			fminsearch(xcs,ximg);
			xtoz(zimg,ximg,h,lb);
			img[i] = ztocell(zimg,N);
		}

		i+=blockDim.x*gridDim.x;
	}
}

//sequential scm search with gr, pe and st arrays
void scm_unravel(unsigned long long *img, long tot_cells, int *gr, int *pe, int*st){
	unsigned long long cell_new,cell_old;
	int i,j,k,m;
	int gr1,pe1,st1;
	int g = 1;
	std::vector<unsigned long long> path;
	bool flag;

	for(i=0;i<tot_cells;i++){
		if(gr[i]!=0 && gr[i]!=-1)
			continue;

		//generate a path from the current virgin cell
		cell_old = i+1;
		flag = true;
		while(flag){
			//process sink cell first
			if(cell_old==0){
				for(j=0;j<path.size();j++){
					gr[path[j]-1] = 1;
					st[path[j]-1] = 0;
					pe[path[j]-1] = 1;
				}
				path.clear();
				break;
			}

			switch(gr[cell_old-1]){
			case 0:
				//virgin cell ahead, keep exploring
				gr[cell_old-1]=-1;
				path.push_back(cell_old);
				cell_new = img[cell_old-1];
				cell_old = cell_new;
				break;
			case -1:
				//cyclic structure, new group found
				g++;
				for(j=path.size()-1,k=1;j>=0;j--,k++){
					if(cell_new!=path[j])
						st[path[j]-1] = 0; //zero step number for cyclic cells
					else
						break;
				}
				for(j=0;j<path.size();j++){
					gr[path[j]-1] = g;
					pe[path[j]-1] = k;
				}
				for(j=path.size()-k-1,m=1;j>=0;j--,m++)
					st[path[j]-1] = m;
				
				path.clear();
				flag = false;
				break;
			default:
				//merge to another group
				gr1 = gr[cell_new-1];
				st1 = st[cell_new-1];
				pe1 = pe[cell_new-1];
				for(j=path.size()-1,k=1;j>=0;j--,k++){
					gr[path[j]-1] = gr1;
					pe[path[j]-1] = pe1;
					st[path[j]-1] = st1+k;
				}
				
				path.clear();
				flag = false;
				break;
			}
		}
	}
}

//extract boundary in cell space from a saddle cell
void boundary(int *gr, int *N, int *bn_old, long tot_cells, unsigned long long cell){
	int *tgt_cells;
	int *bn_new;
	long i,j,k;
	unsigned long long ncells[pop], cs, nc;

	bn_new = new int[tot_cells];
	tgt_cells = new int[tot_cells];
	memset(bn_new,0,tot_cells*sizeof(int));
	memset(tgt_cells,0,tot_cells*sizeof(int));
	
	tgt_cells[cell-1] = 1;
	bn_new[cell-1] = 1;
	bn_old[cell-1] = 1; //bn_old must be initialized with all zeros

	//continuation like extraction
	while(true){
		for(i=0;i<tot_cells;i++){
			if(tgt_cells[i]==0)
				continue;
			cs = i+1;
			neighbour_finder(ncells,N,cs);
			for(j=0;j<pop;j++){
				nc = ncells[j];
				if(nc==0)
					continue;
				if(gr[cs-1]!=gr[nc-1] && bn_old[nc-1]==0){
					//new boundary cell brought in
					tgt_cells[nc-1] = 1;
					bn_new[nc-1] = 1;
				}
			}
			//remove cs from the target cell set since it's already been processed
			tgt_cells[cs-1] = 0;
		}

		//check whether steay state reaches
		for(i=0;i<tot_cells;i++)
			if(bn_new[i]!=bn_old[i])
				break;
		
		if(i==tot_cells)
			break;
		else
			memcpy(bn_old,bn_new,tot_cells*sizeof(int));
	}

	delete[] tgt_cells;
	delete[] bn_new;
}

void saveSCM(int *gr, int *pe, int *st, int *bd, long tot_cells, int*N){
	long i;
	ofstream outData;
	//outData.open("SCMproperties.dat"); //store gr, pe, st
	//for(i=0;i<tot_cells;i++)
	//	outData<<gr[i]<<"   "<<pe[i]<<"   "<<st[i]<<"   "<<bd[i]<<endl;
	//outData.close();
	outData.open("boundary_cells.dat");
	for(i=0;i<tot_cells;i++)
		if(bd[i]==1)
			outData<<i+1<<endl;
	outData.close();
	outData.open("SCMpartition.dat");
	for(i=0;i<D;i++)
		outData<<N[i]<<endl;
	outData.close();
}

void pscm_bd(int *N, float *lb, float *ub){
	/*
	parallel analysis of simple cell mapping, the mapping construction
	is with sop approach, which is implemented in parallel. scm unravelling
	is conducted with Prof. Hsu's traditional sequential approach
	*/
	int *device_N,num_sd,*bd,*bd_new;
	fstream file;
	vector<vector<float>> saddles;
	vector<unsigned long long> mcells;
	vector <float> rowVector(D);
	float *device_lb, *device_ub;
	int *gr,*pe,*st,z[D];
	float x[D],h[D];
	unsigned long long *img, *device_img,*scells;
	long tot_cells=1,i,j;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	clock_t cpu_time_start, cpu_time_end;
	float cpu_time;

	for(i=0;i<D;i++)
		tot_cells*=N[i];

	cout<<"Performing SCM sweeping..."<<endl;

	hipMalloc(&device_N, D*sizeof(int));
	hipMalloc(&device_lb, D*sizeof(float));
	hipMalloc(&device_ub, D*sizeof(float));	
	hipMemcpy(device_N, N, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_lb, lb, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_ub, ub, D*sizeof(int), hipMemcpyHostToDevice);

	//---------------------------build scm in parallel------------------------------------
	hipMalloc(&device_img, tot_cells*sizeof(unsigned long long));
	scm_sop<<<NumberOfBlock,ThreadPerBlock>>>(tot_cells,device_N,device_lb,device_ub,device_img);
	hipDeviceSynchronize();
	img = new unsigned long long[tot_cells];
	hipMemcpy(img,device_img,tot_cells*sizeof(unsigned long long),hipMemcpyDeviceToHost);
	hipFree(device_img);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float gpu_time;
	hipEventElapsedTime(&gpu_time, start, stop);

	//------------------------sequential scm unravelling----------------------------------
	cpu_time_start = clock();
	gr = new int[tot_cells];
	pe = new int[tot_cells];
	st = new int[tot_cells];
	memset(gr,0,tot_cells*sizeof(int));
	memset(pe,0,tot_cells*sizeof(int));
	memset(st,0,tot_cells*sizeof(int));
	scm_unravel(img,tot_cells,gr,pe,st);

	//-------------------------extract boundary from scm----------------------------------
	//read the saddle points from matlab generated file
	//read cells captured by ga from the file
	num_sd = 0;
	file.open("saddles.dat");
	if(file.is_open()){
		while(file.good()){
			saddles.push_back(rowVector);
			for(i=0;i<D;i++)
				file >> saddles[num_sd][i];
			num_sd++;
		}
	}
	saddles.erase(saddles.end()-1);
	
	num_sd--;
	scells = (unsigned long long*)malloc(num_sd*sizeof(unsigned long long));
	for(i=0;i<D;i++)
		h[i] = (ub[i]-lb[i])/N[i];
	for(i=0;i<num_sd;i++){
		for(j=0;j<D;j++)
			x[j] = saddles[i][j];
		xtoz(z,x,h,lb);
		scells[i] = ztocell(z,N);
	}
	saddles.clear();

	cout<<endl;
	cout<<"Extracting boundaries..."<<endl;
	bd = new int[tot_cells];
	bd_new = new int[tot_cells];
	memset(bd,0,tot_cells*sizeof(int));
	memset(bd_new,0,tot_cells*sizeof(int));
	
	//find boundary start from all saddle cells
	for(i=0;i<num_sd;i++){
		boundary(gr,N,bd_new,tot_cells,scells[i]);
		//elementwise operation
		for(j=0;j<tot_cells;j++)
			if(bd_new[j]==1 && bd[j]==0)
				bd[j] = 1;
		memset(bd_new,0,tot_cells*sizeof(int));
	}

	cpu_time_end = clock();
	cpu_time = float(cpu_time_end-cpu_time_start)/CLOCKS_PER_SEC*1000;
	cout<<endl;
	cout<<"Total SCM analysis runtime is: "<<setw(8)<<gpu_time+cpu_time<<setprecision(6)<<" ms"<<endl;

	//---------------save global properties for post processing-------------------
	saveSCM(gr,pe,st,bd,tot_cells,N);

	delete[] bd;
	delete[] bd_new;
	delete[] scells;
	delete[] img;
	delete[] gr;
	delete[] pe;
	delete[] st;

	hipFree(device_N);
	hipFree(device_lb);
	hipFree(device_ub);
}

int main(){
	int Nscm[D] = {120,120,120};
	float lb[D] = {-2.5f,-2.5f,-2.5f};
	float ub[D] = {2.5f,2.5f,2.5f};

	pscm_bd(Nscm,lb,ub); //pure scm for global analysis on boundary finding

	return 0;
}