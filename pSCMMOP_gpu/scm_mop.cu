#include "hip/hip_runtime.h"
/*
CUDA kernels for hybrid scm-mop on gpu
Free Xiong
2014/02/27
*/
#include "hip/hip_runtime.h"
#include <cstdlib>
#include "cell.cuh"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <stdio.h>

using namespace std;

#define NumberOfBlock 2048
#define ThreadPerBlock 1024

//evaluate cells of central points and function values
__global__ void cellEvaluation(const int * cells, const float * lb, const float * ub, 
									 const int * N, const int tot_cells, float * xc, float * fe){
	/*
	Input arguments:
	     cells:    cells waiting to be evaluated
		 lb,ub:    lower and upper bound of searching domain
		   N:      celluar space partition
	   tot_cells:  total number of cells in "cells" array

	Output arguments:
	      xc:      1D array with cell central coordinates
		  fe:      1D array with central function values
	*/
	int id = threadIdx.x + blockIdx.x*blockDim.x; //unique id of thread
	int i,j;
	int z[D];
	int cs;
	float h[D];
	float xd[D];
	float fd[O];
	
	for(i=0;i!=D;i++)
		h[i] = (ub[i]-lb[i])/N[i];

	i = id;
	while(i<tot_cells){
		cs = cells[i];
		celltoz(z,cs,N);
		ztox(xd,z,h,lb);
		f(fd,xd);
		for(j=0;j!=D;j++)
			xc[i*D+j]=xd[j];
		for(j=0;j!=O;j++)
			fe[i*O+j]=fd[j];
		i+=blockDim.x*gridDim.x;
	}
}

//creat simple cell mapping
__global__ void scm_ds(const int* cells, const int tot_cells, const float * xc, 
					   const float * fe, const float* lb, const float* ub, const int* N, 
					   int* img){
	//scm with directed search
	/*
	Input arguments:
	cells:     input cell set
	tot_cells: number of cells in the set
	lb,ub:     lower and upper bound
	N:         cell space partition
	xc,fe:     cell info evalauted outside

	Output argument:
	img:      image cells
	*/

	//scm
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int i,j,k,cs;
	bool flag,better,dominated,constraints;
	int index;
	float xcs[D],xn[D],fc[O],fn[O],cst[C];
	float cmp,df_old,df_new;
	int neighbour[pop];
	//int z_c[D], z_fwd[D], z_bwk[D];

	//for(i=0;i<D;i++)
	//	pop*=3;

	// loop over the entire cell set
	i = id;
	while(i<tot_cells){
		cs = cells[i];
		for(j=0;j<D;j++)
			xcs[j] = xc[i*D+j];// central cell coordinates
		for(j=0;j<O;j++)
			fc[j] = fe[i*O+j];// central cell function values
		constraints = true;

		//check central cell constaints violation
		g(cst,xcs);
		for(j=0;j<C;j++){
			if(cst[j]>0){
				constraints = false;
				break;
			}
		}
		if(!constraints){
			img[i] = 0; //sink cell
			i+=blockDim.x*gridDim.x;
			continue;
		}

		neighbour_finder(neighbour,N,cs); //be aware of the visitation of neighbour array

		////search among neighbours that are along the coordinate directions (2*dim)
		//celltoz(z_c,cs,N);
		//for(j=0;j<D;j++){
		//	z_bwk[j] = z_c[j];
		//	z_fwd[j] = z_c[j];
		//}
		//for(j=0;j<D;j++){
		//	z_fwd[j]+=1; //increase along the jth direction
		//	z_bwk[j]-=1;
		//	if(z_fwd[j]>N[j])
		//		neighbour[j] = 0; //out of upper boundary
		//	else
		//		neighbour[j] = ztocell(z_fwd,N);
		//	if(z_bwk[j]<1)
		//		neighbour[j+D] = 0; //out of lower boundary
		//	else
		//		neighbour[j+D] = ztocell(z_bwk,N);
		//	z_fwd[j] = z_c[j]; //recover the coordinate for one direction increasing only
		//	z_bwk[j] = z_c[j];
		//}
		
		df_old = -1.0f;
		dominated = false;
		for(j=0;j<pop;j++){
			//check for valid neighour and compare
			flag = false;
			for(k=0;k<tot_cells;k++){
				if(cells[k]==neighbour[j]){
					flag = true;
					index = k;
					break;
				}
			}
			if(!flag)
				index = -1;
			
			//dominance optimality when picking the image cell
			if(neighbour[j]==0 || !flag)
				continue;
			else{
				for(k=0;k<O;k++)
					fn[k] = fe[index*O+k]; //neighbour function values
				
				//all(fn<fc)
				cmp = 10.0f;
				better = true;
				for(k=0;k<O;k++){
					cmp = fn[k]-fc[k];
					if(cmp>0.0f){
						better = false;
						break;
					}
				}

				//steepest descent
				if(better){
					dominated = true; //there is better neighbour
					df_new = 0.0f;
					for(k=0;k<O;k++)
						df_new+=fc[k]-fn[k];
					if(df_new>df_old){
						img[i] = neighbour[j];
						df_old = df_new; //swap for next df comparison
					}
					//check violation of constraints of the neighbours
					for(k=0;k<D;k++)
						xn[k] = xc[index*D+k]; //neighbour center
					g(cst,xn);
					for(k=0;k<C;k++){
						if(cst[k]>0){
							img[i] = 0; //constraints violation, sink cell
							break;
						}
					}
				}
			}
		}

		if(!dominated)
			img[i] = cs;

		i+=blockDim.x*gridDim.x;
	}
}

//scm unravelling for p-k cells
__global__ void scm_search(const int * cells, const int * img, const int tot_cells, 
						   const int max_period, int * period){
	//scm search for periodic cells
	/*
	Input arguments:
	cells:        cell set
	img:          image of cells
	tot_cells:    total number of cell set
	max_period:   maximum period extracted, mostly p-2

	Ouput argument:
	period:       period of each cell, transients cell has -1 period
	*/
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int i,j,k,cs,cell_old,cell_new,index;
	bool pkcell;
	
	//loop over entire cell set
	i = id;
	while(i<tot_cells){
		cs = cells[i];
		cell_old = cs;
		pkcell = false;

		//shoot forward to extract cyclic structures
		for(j=0;j<max_period+1;j++){
			//find index
			for(k=0;k<tot_cells;k++){
				if(cells[k]==cell_old){
					index = k;
					break;
				}
			}
			//shoot forward
			cell_new = img[index];
			if(cell_new==0){
				period[i] = -1; // meet sink cell
				break;
			}
			else if(cell_new==cs){
				period[i] = j+1;
				pkcell = true;
				break;
			}
			cell_old = cell_new;
		}

		//mark transient cells
		if(!pkcell)
			period[i] = -1;

		i+=blockDim.x*gridDim.x;
	}
}

//parallel refinement
__global__ void refine(int * rcells, const int * cells, const int tot_cells, int * N_new,
					   const int * N, const int * div, const int * allcomb, const int dim){
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int i,cs;
	i = id;

	//loop over candidate cell set
	while(i<tot_cells){
		cs = cells[i];
		subdivision(&rcells[dim*i],cs,N,N_new,dim,div,allcomb);
		i+=blockDim.x*gridDim.x;
	}
}

//parallel dominance check
__global__ void dominance(const float * fe, const int * period, const int total_cells, int * flag){
	//dominance check over the candidate cells
	/*
	Input arguments:
	fe:       function values of all cells
	period:   periodic number of all cells
	tot_cells:  total number of cell set

	Output argument:
	flag:     indicator of whether a cell is a Pareto cell (0,1)
	*/
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int i,j,k;
	float fc[O],cmp[O];
	int temp[O],gz;
	bool dominated, allequal;

	//loop over entire cell set
	i = id;
	while(i<total_cells){
		dominated = false;
		if(period[i]==-1){
			flag[i] = 0;
			i+=blockDim.x*gridDim.x;
			continue;
		}
		else{
			for(j=0;j<O;j++)
				fc[j] = fe[i*O+j];
			//compare function values with other candidates
			for(j=0;j<total_cells;j++){
				if(period[j]==-1)
					continue;
				else{
					allequal = true;
					//get the function value difference and compare (temp=1 is bad)
					for(k=0;k<O;k++){
						cmp[k] = fc[k]-fe[j*O+k];
						if(cmp[k]>=0)
							temp[k]=1;
						else
							temp[k]=0;
					}
					//check if all objectives are equal happens
					for(k=0;k<O;k++){
						if(cmp[k]!=0){
							allequal = false;
							break;
						}
					}
					//check whether (all(cmp>=0) && !all(cmp==0))  stands
					gz=1;
					for(k=0;k<O;k++)
						gz*=temp[k];
					if(gz==1 && !allequal){
						dominated = true;
						flag[i] = 0;
						break;
					}
				}
			}
		}
		if(!dominated)
			flag[i] = 1;

		i+=blockDim.x*gridDim.x;
	}
}

void saveData(float * xc, float * fe, int write_cells, int * status){
	//save final Pareto set and front into different files
	int i,j;
	ofstream outData;
	int counter=0;

	outData.open("ps.dat");
	for(i=0;i<write_cells;i++){
		if(status[i]==1){
			counter++;
			for(j=0;j<D;j++)
				outData<<scientific<<setw(15)<<xc[i*D+j]<<setprecision(4);
			outData<<endl;
		}
	}
	outData.close();

	outData.open("pf.dat");
	for(i=0;i<write_cells;i++){
		if(status[i]==1){
			for(j=0;j<O;j++)
				outData<<scientific<<setw(15)<<fe[i*O+j]<<setprecision(4);
			outData<<endl;
		}
	}
	outData.close();
	
	cout<<endl;
	cout<<"Number of Pareto cells is: "<<counter<<endl;
}

void scm_mop(int * N, float * lb, float * ub, int * div, int max_period, int max_iter){
	/*
	main structure of scm-mop hybrid algorithm, this code execute kernels
	from host (CPU) to device (GPU)
	*/
	int i,j,iter,tot_cells=1,dim=1;
	int write_cells;
	int * cells;
	float * device_xc, * device_fe;
	int * device_N, * device_div;
	float * device_lb, * device_ub;
	int * device_cells, * device_img, * device_period;
	int * device_candcells;
	int candNo;
	int * device_Nnew, * device_rcells;
	int * allcomb, * device_allcomb;
	int * device_flag;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for(i=0;i<D;i++){
		tot_cells *= N[i];
		dim *= 2*div[i]+1;
	}

	allcomb = new int [dim*D];
	cartprod(allcomb,div); //cartesin product of cell coord increasing
	
	hipMalloc(&device_N, D*sizeof(int));
	hipMalloc(&device_div, D*sizeof(int));
	hipMalloc(&device_lb, D*sizeof(float));
	hipMalloc(&device_ub, D*sizeof(float));
	hipMalloc(&device_allcomb, dim*D*sizeof(int));
	hipMalloc(&device_Nnew, D*sizeof(int));

	hipMemcpy(device_N, N, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_div, div, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_lb, lb, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_ub, ub, D*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_allcomb, allcomb, D*dim*sizeof(int), hipMemcpyHostToDevice);

	iter = 0;

//--start of the loop
	while(iter<max_iter){
		//intitial cell set
		if(iter==0){
			cells = new int[tot_cells];
			for(i=0;i<tot_cells;i++)
				cells[i] = i+1; //1-base cell number counting
			hipMalloc(&device_cells, tot_cells*sizeof(int));
			hipMemcpy(device_cells, cells, tot_cells*sizeof(int), hipMemcpyHostToDevice);
			delete[] cells;
		}

		write_cells = tot_cells;

		//evaluate cells
		hipMalloc(&device_xc, tot_cells*D*sizeof(float));
		hipMalloc(&device_fe, tot_cells*O*sizeof(float));
		cellEvaluation<<<NumberOfBlock,ThreadPerBlock>>>(device_cells,device_lb,device_ub,device_N,tot_cells,device_xc,device_fe);
		hipDeviceSynchronize();

		//build scm
		hipMalloc(&device_img, tot_cells*sizeof(int));
		scm_ds<<<NumberOfBlock,ThreadPerBlock>>>(device_cells,tot_cells,device_xc,device_fe,device_lb,device_ub,device_N,device_img);
		hipDeviceSynchronize();
		//if(iter==0){
		//	int * temp = new int [tot_cells];
		//	hipMemcpy(temp, device_cells, tot_cells*sizeof(int), hipMemcpyDeviceToHost);
		//	for(i=0;i<100;i++){
		//			cout<<setw(5)<<temp[i];
		//		cout<<endl;
		//	}
		//	delete[] temp;
		//	//system("pause");
		//}

		//unravel scm
		hipMalloc(&device_period, tot_cells*sizeof(int));
		scm_search<<<NumberOfBlock,ThreadPerBlock>>>(device_cells,device_img,tot_cells,max_period,device_period);
		hipDeviceSynchronize();

		//collect candidate cells on CPU
		int * period = new int [tot_cells];
		hipMemcpy(period, device_period, tot_cells*sizeof(int), hipMemcpyDeviceToHost);
		candNo = 0;
		for(i=0;i<tot_cells;i++){
			if(period[i]!=-1)
				candNo++;
		}
		cout<<"Iteration "<<iter+1<<" complete. ";
		cout<<candNo<<" cells found..."<<endl;

		int * candcells = new int [candNo];
		j=0;
		cells = new int [tot_cells];
		hipMemcpy(cells, device_cells, tot_cells*sizeof(int), hipMemcpyDeviceToHost);
		for(i=0;i<tot_cells;i++){
			if(period[i]!=-1){
				candcells[j] = cells[i];
				j++;
			}
		}
		delete[] cells;

		hipMalloc(&device_candcells, candNo*sizeof(int));
		hipMemcpy(device_candcells, candcells, candNo*sizeof(int), hipMemcpyHostToDevice);
		delete[] candcells;
		delete[] period;
	
		//refine
		if(iter<max_iter-1){
			hipMalloc(&device_rcells, candNo*dim*sizeof(int));
			refine<<<NumberOfBlock,ThreadPerBlock>>>(device_rcells,device_candcells,candNo,device_Nnew,device_N,device_div,device_allcomb,dim);
			hipDeviceSynchronize();

			//ready for next iteration
			tot_cells = candNo*dim;
			hipFree(device_cells);
			hipMalloc(&device_cells, candNo*dim*sizeof(int));
			hipMemcpy(device_cells, device_rcells, candNo*dim*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(device_N, device_Nnew, D*sizeof(int), hipMemcpyDeviceToDevice);

			hipFree(device_rcells);
			hipFree(device_period);
			hipFree(device_xc);
			hipFree(device_fe);
		}

		iter++;

		hipFree(device_candcells);
		hipFree(device_img);
	}
//--end of the loop

	//dominance check
	hipMalloc(&device_flag, write_cells*sizeof(int));
	dominance<<<NumberOfBlock,ThreadPerBlock>>>(device_fe,device_period,write_cells,device_flag);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	//write Pareto results to file
	float * xc = new float [write_cells*D];
	float * fe = new float [write_cells*O];
	int * flag = new int [write_cells];
	hipMemcpy(flag, device_flag, write_cells*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(xc, device_xc, write_cells*D*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fe, device_fe, write_cells*O*sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float gpu_time;
	hipEventElapsedTime(&gpu_time, start, stop);
	cout<<endl;
	cout<<"Device (GPU) runtime is: "<<setw(8)<<gpu_time<<setprecision(6)<<" ms"<<endl;

	saveData(xc,fe,write_cells,flag);

	//free up GPU memory
	hipFree(device_N);
	hipFree(device_div);
	hipFree(device_lb);
	hipFree(device_ub);
	hipFree(device_allcomb);
	hipFree(device_Nnew);
}

int main(){
	int N[D] = {7,7,7,7,7};
	float lb[D] = {0,0,0,0,0};
	float ub[D] = {40,40,40,40,40};
	int div[D] = {1,1,1,1,1};
	int max_iter = 2;
	int max_period = 2;

	scm_mop(N,lb,ub,div,max_period,max_iter); //GPU implementation

	return 0;
}